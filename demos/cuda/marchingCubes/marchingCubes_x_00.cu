#include "hip/hip_runtime.h"

/*
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/* 
    Marching cubes

    This sample extracts a geometric isosurface from a volume dataset using
    the marching cubes algorithm. It uses the scan (prefix sum) function from
    the Thrust library to perform stream compaction.  Similar techniques can
    be used for other problems that require a variable-sized output per
    thread.

    For more information on marching cubes see:
    http://local.wasp.uwa.edu.au/~pbourke/geometry/polygonise/
    http://en.wikipedia.org/wiki/Marching_cubes

    Volume data courtesy:
    http://www9.informatik.uni-erlangen.de/External/vollib/

    For more information on the Thrust library
    http://code.google.com/p/thrust/

    The algorithm consists of several stages:

    1. Execute "classifyVoxel" kernel
    This evaluates the volume at the corners of each voxel and computes the
    number of vertices each voxel will generate.
    It is executed using one thread per voxel.
    It writes two arrays - voxelOccupied and voxelVertices to global memory.
    voxelOccupied is a flag indicating if the voxel is non-empty.

    2. Scan "voxelOccupied" array (using Thrust scan)
    Read back the total number of occupied voxels from GPU to CPU.
    This is the sum of the last value of the exclusive scan and the last
    input value.

    3. Execute "compactVoxels" kernel
    This compacts the voxelOccupied array to get rid of empty voxels.
    This allows us to run the complex "generateTriangles" kernel on only
    the occupied voxels.

    4. Scan voxelVertices array
    This gives the start address for the vertex data for each voxel.
    We read back the total number of vertices generated from GPU to CPU.

    Note that by using a custom scan function we could combine the above two
    scan operations above into a single operation.

    5. Execute "generateTriangles" kernel
    This runs only on the occupied voxels.
    It looks up the field values again and generates the triangle data,
    using the results of the scan to write the output to the correct addresses.
    The marching cubes look-up tables are stored in 1D textures.

    6. Render geometry
    Using number of vertices from readback.
*/

// std includes
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>


//#include "cutil_math.h"

#include <cutil_inline.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h
#include <shrQATest.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>
#include <vector_functions.h>


/* #include <GL/glew.h> */
/* #include <cutil_inline.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h */
/* #include <cutil_gl_inline.h> */
/* #include <shrQATest.h> */
/* #include <hip/hip_runtime_api.h> */
/* #include <cuda_gl_interop.h> */
/* #include <rendercheck_gl.h> */
/* #include <hip/hip_vector_types.h> */
/* #include <vector_functions.h> */

#include "defines.h"
#include "graphics.hpp"
#include "io.hpp"

#define MAX_EPSILON_ERROR 5.0f
#define REFRESH_DELAY	  10 //ms
#define MAX(a,b) ((a > b) ? a : b)
#define EPSILON 5.0f
#define THRESHOLD 0.30f


//KERNEL FUNCTIONS (DEFINED IN marchingCubes_kernel.cu)
extern "C" void
launch_classifyVoxel( dim3 grid, dim3 threads, uint* voxelVerts, uint *voxelOccupied, uchar *volume,
					  uint3 gridSize, uint3 gridSizeShift, uint3 gridSizeMask, uint numVoxels,
					  float3 voxelSize, float isoValue);

extern "C" void 
launch_compactVoxels(dim3 grid, dim3 threads, uint *compactedVoxelArray, uint *voxelOccupied, 
					uint *voxelOccupiedScan, uint numVoxels);

extern "C" void
launch_generateTriangles(dim3 grid, dim3 threads,
						float4 *pos, float4 *norm, uint *compactedVoxelArray, uint *numVertsScanned,
						uint3 gridSize, uint3 gridSizeShift, uint3 gridSizeMask,
						float3 voxelSize, float isoValue, uint activeVoxels, uint maxVerts);

extern "C" void
launch_generateTriangles2(dim3 grid, dim3 threads,
						float4 *pos, float4 *norm, uint *compactedVoxelArray, uint *numVertsScanned, uchar *volume,
						uint3 gridSize, uint3 gridSizeShift, uint3 gridSizeMask,
						float3 voxelSize, float isoValue, uint activeVoxels, uint maxVerts);

extern "C" void allocateTextures(	uint **d_edgeTable, uint **d_triTable,  uint **d_numVertsTable );
extern "C" void bindVolumeTexture(uchar *d_volume);
extern "C" void ThrustScanWrapper(unsigned int* output, unsigned int* input, unsigned int numElements);


// Auto-Verification Code
/* const int frameCheckNumber = 4; */
/* int fpsCount = 0;        // FPS count for averaging */
/* int fpsLimit = 1;        // FPS limit for sampling */
/* int g_Index = 0; */
/* unsigned int frameCount = 0; */
/* unsigned int g_TotalErrors = 0; */
/* bool g_Verify = false; */
/* bool g_bQAReadback = false; */
/* bool g_bOpenGLQA   = false; */
/* bool g_bFBODisplay = false; */

/* int *pArgc = NULL; */
/* char **pArgv = NULL; */

// CheckFBO/BackBuffer class objects
/* CFrameBufferObject  *g_FrameBufferObject = NULL; */
/* CheckRender         *g_CheckRender       = NULL; */

/* template <class T> */
/* void dumpBuffer(T *d_buffer, int nelements, int size_element); */

/* template <class T> */
/* void dumpFile(T *d_buffer, int nelements, int size_element, const char *filename); */




struct MarchingCubesProgram {

    MarchingCubesProgram(){}

    ~MarchingCubesProgram(){
      cleanup();
      reset();
      exit();
    }

      bool bWireframe;
      float3 rotate;
      float3 translate;

    //member variables
      unsigned int timer;

      const char *volumeFilename ;

      uint3 gridSizeLog2;// = make_uint3(5, 5, 5);
      uint3 gridSizeShift;
      uint3 gridSize;
      uint3 gridSizeMask;

      float3 voxelSize;
      uint numVoxels    ;
      uint maxVerts     ;
      uint activeVoxels ;
      uint totalVerts   ;

      float isoValue	  ;
      float dIsoValue		;

      // device data
      GLuint posVbo, normalVbo;
      GLint  gl_Shader;
      struct hipGraphicsResource *cuda_posvbo_resource, *cuda_normalvbo_resource; // handles OpenGL-CUDA exchange

      float4 *d_pos, *d_normal;

      uchar *d_volume ;
      uint *d_voxelVerts ;
      uint *d_voxelVertsScan ;
      uint *d_voxelOccupied ;
      uint *d_voxelOccupiedScan ;
      uint *d_compVoxelArray;

      // tables
      uint* d_numVertsTable;
      uint* d_edgeTable;
      uint* d_triTable;


      CheckRender         *g_CheckRender;


    ////////////////////////////////////////////////////////////////////////////////
    // initialize marching cubes
    ////////////////////////////////////////////////////////////////////////////////
    void init(int argc, char** argv)
    {

        timer = 0;
        volumeFilename = "Bucky.raw";

        numVoxels    = 0;
        maxVerts     = 0;
        activeVoxels = 0;
        totalVerts   = 0;

        isoValue		= 0.2f;
        dIsoValue		= 0.005f;

        d_pos = 0; d_normal = 0; d_volume = 0; d_voxelVerts = 0; d_voxelVertsScan=0;
        d_voxelOccupied=0;  d_voxelOccupiedScan=0;  d_compVoxelArray=0;

        d_numVertsTable=0; d_edgeTable=0; d_triTable=0;

        gridSizeLog2 = make_uint3(5, 5, 5);
        gridSize = make_uint3(1<<gridSizeLog2.x, 1<<gridSizeLog2.y, 1<<gridSizeLog2.z);
        gridSizeMask = make_uint3(gridSize.x-1, gridSize.y-1, gridSize.z-1);
        gridSizeShift = make_uint3(0, gridSizeLog2.x, gridSizeLog2.x+gridSizeLog2.y);

        numVoxels = gridSize.x*gridSize.y*gridSize.z;
        voxelSize = make_float3(2.0f / gridSize.x, 2.0f / gridSize.y, 2.0f / gridSize.z);
        maxVerts = gridSize.x*gridSize.y*100;

        bWireframe = false;
        rotate = make_float3(0.0, 0.0, 0.0);
        translate= make_float3(0.0, 0.0, -3.0);

        g_CheckRender = NULL;


        printf("grid: %d x %d x %d = %d voxels\n", gridSize.x, gridSize.y, gridSize.z, numVoxels);
        printf("max verts = %d\n", maxVerts);

#if SAMPLE_VOLUME
        // load volume data
        char* path = cutFindFilePath(volumeFilename, argv[0]);
        if (path == NULL) {
            fprintf(stderr, "Error finding file '%s'\n", volumeFilename);
            cutilDeviceReset();
            shrQAFinishExit(argc, (const char **)argv, QA_FAILED);
        }

        int size = gridSize.x*gridSize.y*gridSize.z*sizeof(uchar);
        uchar *volume = loadRawFile(path, size);
        cutilSafeCall(hipMalloc((void**) &d_volume, size));
        cutilSafeCall(hipMemcpy(d_volume, volume, size, hipMemcpyHostToDevice) );
        free(volume);

      bindVolumeTexture(d_volume);
#endif

        // create VBOs
        createVBO(&posVbo, maxVerts*sizeof(float)*4);
        cutilSafeCall(hipGraphicsGLRegisterBuffer(&cuda_posvbo_resource, posVbo, 
        cudaGraphicsMapFlagsWriteDiscard));

        createVBO(&normalVbo, maxVerts*sizeof(float)*4);
        cutilSafeCall(hipGraphicsGLRegisterBuffer(&cuda_normalvbo_resource, normalVbo, 
        cudaGraphicsMapFlagsWriteDiscard));

        // allocate textures
        allocateTextures(	&d_edgeTable, &d_triTable, &d_numVertsTable );

        // allocate device memory
        unsigned int memSize = sizeof(uint) * numVoxels;
        cutilSafeCall(hipMalloc((void**) &d_voxelVerts,            memSize));
        cutilSafeCall(hipMalloc((void**) &d_voxelVertsScan,        memSize));
        cutilSafeCall(hipMalloc((void**) &d_voxelOccupied,         memSize));
        cutilSafeCall(hipMalloc((void**) &d_voxelOccupiedScan,     memSize));
        cutilSafeCall(hipMalloc((void**) &d_compVoxelArray,   memSize));
    }

    
    void cleanup()
    {
        cutilCheckError( cutDeleteTimer( timer ));

        deleteVBO(&posVbo,    &cuda_posvbo_resource);
        deleteVBO(&normalVbo, &cuda_normalvbo_resource);

        cutilSafeCall(hipFree(d_edgeTable));
        cutilSafeCall(hipFree(d_triTable));
        cutilSafeCall(hipFree(d_numVertsTable));

        cutilSafeCall(hipFree(d_voxelVerts));
        cutilSafeCall(hipFree(d_voxelVertsScan));
        cutilSafeCall(hipFree(d_voxelOccupied));
        cutilSafeCall(hipFree(d_voxelOccupiedScan));
        cutilSafeCall(hipFree(d_compVoxelArray));

        if (d_volume) cutilSafeCall(hipFree(d_volume));
    }


    void
    run(int argc, char** argv)
    {
       printf("MarchingCubes ");
       
       shrQAStart(argc, argv); 
       cudaGLSetGLDevice( cutGetMaxGflopsDeviceId() ); 
       init(argc, argv);
       cutilCheckError( cutCreateTimer( &timer));

      //now run GL loop
   }

    void start(int argc, char** argv){ shrQAStart(argc, argv); }
    void reset(){ cutilDeviceReset(); }
    void exit(int argc=0, char** argv=NULL) { shrQAFinishExit(argc, (const char **)argv, QA_PASSED); }


    //DEBUGGING      
#define DEBUG_BUFFERS 0

    /* template <class T> */
    /* void dumpBuffer(T *d_buffer, int nelements, int size_element) */
    /* { */
    /*     uint bytes = nelements * size_element; */
    /*     T *h_buffer = (T *) malloc(bytes); */
    /*     cutilSafeCall( hipMemcpy(h_buffer, d_buffer, bytes, hipMemcpyDeviceToHost) ); */
    /*     for(int i=0; i<nelements; i++) { */
    /*         printf("%d: %u\n", i, h_buffer[i]); */
    /*     } */
    /*     printf("\n"); */
    /*     free(h_buffer); */
    /* } */

    /* template <class T> */
    /* void dumpFile(T *d_buffer, int nelements, int size_element, const char *filename) */
    /* { */
    /*     cutilSafeCall( hipMemcpy( (T *)g_CheckRender->imageData(), (T *)d_buffer, nelements*size_element, hipMemcpyDeviceToHost) ); */
    /*     g_CheckRender->dumpBin((unsigned char *)g_CheckRender->imageData(), nelements*size_element, filename); */
    /* } */

    ////////////////////////////////////////////////////////////////////////////////
    //! Run the Cuda part of the computation
    ////////////////////////////////////////////////////////////////////////////////
      void computeIsosurface()
      {
          int threads = 128;
          dim3 grid(numVoxels / threads, 1, 1);
          // get around maximum grid size of 65535 in each dimension
          if (grid.x > 65535) {
              grid.y = grid.x / 32768;
              grid.x = 32768;
          }

          // calculate number of vertices need per voxel
          launch_classifyVoxel(grid, threads, 
                  d_voxelVerts, d_voxelOccupied, d_volume, 
                  gridSize, gridSizeShift, gridSizeMask, 
                  numVoxels, voxelSize, isoValue);
#if DEBUG_BUFFERS
          printf("voxelVerts:\n");
          dumpBuffer(d_voxelVerts, numVoxels, sizeof(uint));
#endif

#if SKIP_EMPTY_VOXELS
          // scan voxel occupied array
          ThrustScanWrapper(d_voxelOccupiedScan, d_voxelOccupied, numVoxels);

#if DEBUG_BUFFERS
          printf("voxelOccupiedScan:\n");
          dumpBuffer(d_voxelOccupiedScan, numVoxels, sizeof(uint));
#endif

          // read back values to calculate total number of non-empty voxels
          // since we are using an exclusive scan, the total is the last value of
          // the scan result plus the last value in the input array
          {
              uint lastElement, lastScanElement;
              cutilSafeCall(hipMemcpy((void *) &lastElement, 
                             (void *) (d_voxelOccupied + numVoxels-1), 
                             sizeof(uint), hipMemcpyDeviceToHost));
              cutilSafeCall(hipMemcpy((void *) &lastScanElement, 
                             (void *) (d_voxelOccupiedScan + numVoxels-1), 
                             sizeof(uint), hipMemcpyDeviceToHost));
              activeVoxels = lastElement + lastScanElement;
          }

          if (activeVoxels==0) {
              // return if there are no full voxels
              totalVerts = 0;
              return;
          }

          // compact voxel index array
          launch_compactVoxels(grid, threads, d_compVoxelArray, d_voxelOccupied, d_voxelOccupiedScan, numVoxels);
          cutilCheckMsg("compactVoxels failed");

#endif // SKIP_EMPTY_VOXELS

          // scan voxel vertex count array
          ThrustScanWrapper(d_voxelVertsScan, d_voxelVerts, numVoxels);

#if DEBUG_BUFFERS
          printf("voxelVertsScan:\n");
          dumpBuffer(d_voxelVertsScan, numVoxels, sizeof(uint));
#endif

          // readback total number of vertices
          {
              uint lastElement, lastScanElement;
              cutilSafeCall(hipMemcpy((void *) &lastElement, 
                             (void *) (d_voxelVerts + numVoxels-1), 
                             sizeof(uint), hipMemcpyDeviceToHost));
              cutilSafeCall(hipMemcpy((void *) &lastScanElement, 
                             (void *) (d_voxelVertsScan + numVoxels-1), 
                             sizeof(uint), hipMemcpyDeviceToHost));
              totalVerts = lastElement + lastScanElement;
          }

          // generate triangles, writing to vertex buffers
          size_t num_bytes;
            // DEPRECATED: cutilSafeCall(cudaGLMapBufferObject((void**)&d_pos, posVbo));
            cutilSafeCall(hipGraphicsMapResources(1, &cuda_posvbo_resource, 0));
            cutilSafeCall(hipGraphicsResourceGetMappedPointer((void**)&d_pos, &num_bytes, cuda_posvbo_resource));

            // DEPRECATED: cutilSafeCall(cudaGLMapBufferObject((void**)&d_normal, normalVbo));
            cutilSafeCall(hipGraphicsMapResources(1, &cuda_normalvbo_resource, 0));
            cutilSafeCall(hipGraphicsResourceGetMappedPointer((void**)&d_normal, &num_bytes, cuda_normalvbo_resource));

#if SKIP_EMPTY_VOXELS
          dim3 grid2((int) ceil(activeVoxels / (float) NTHREADS), 1, 1);
#else
          dim3 grid2((int) ceil(numVoxels / (float) NTHREADS), 1, 1);
#endif
          while(grid2.x > 65535) {
              grid2.x/=2;
              grid2.y*=2;
          }
#if SAMPLE_VOLUME
          launch_generateTriangles2(grid2, NTHREADS, d_pos, d_normal, 
                                                  d_compVoxelArray, 
                                                  d_voxelVertsScan, d_volume, 
                                                  gridSize, gridSizeShift, gridSizeMask, 
                                                  voxelSize, isoValue, activeVoxels, 
                                                  maxVerts);
#else
          launch_generateTriangles(grid2, NTHREADS, d_pos, d_normal, 
                                                 d_compVoxelArray, 
                                                 d_voxelVertsScan, 
                                                 gridSize, gridSizeShift, gridSizeMask, 
                                                 voxelSize, isoValue, activeVoxels, 
                                                 maxVerts);
#endif

          cutilSafeCall(hipGraphicsUnmapResources(1, &cuda_normalvbo_resource, 0));
          cutilSafeCall(hipGraphicsUnmapResources(1, &cuda_posvbo_resource, 0));
      }


      ////////////////////////////////////////////////////////////////////////////////
      //! Display callback
      ////////////////////////////////////////////////////////////////////////////////
      void onFrame()
      {
          //cutilCheckError(cutStartTimer(timer));  
          computeIsosurface();

          // Common display code path
        {
          glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

          // set view matrix
          glMatrixMode(GL_MODELVIEW);
          glLoadIdentity();
          glTranslatef(translate.x, translate.y, translate.z);
          glRotatef(rotate.x, 1.0, 0.0, 0.0);
          glRotatef(rotate.y, 0.0, 1.0, 0.0);

          glPolygonMode(GL_FRONT_AND_BACK, bWireframe? GL_LINE : GL_FILL);
          //if (lighting) {
            glEnable(GL_LIGHTING);
          //}

          // render
          //if (render) {
            glPushMatrix();
            glRotatef(180.0, 0.0, 1.0, 0.0);
            glRotatef(90.0, 1.0, 0.0, 0.0);
            renderIsosurface(totalVerts);
            glPopMatrix();
          //}

          glDisable(GL_LIGHTING);
        } 

         // cutilCheckError(cutStopTimer(timer));  

         // computeFPS();

       }


};
